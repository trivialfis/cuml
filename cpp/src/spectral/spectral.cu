#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cuml/manifold/common.hpp>
#include <raft/sparse/coo.cuh>

#include <raft/sparse/linalg/spectral.cuh>

namespace raft {
class handle_t;
}

namespace ML {

namespace Spectral {

/**
   * Given a COO formatted (symmetric) knn graph, this function
   * computes the spectral embeddings (lowest n_components
   * eigenvectors), using Lanczos min cut algorithm.
   * @param rows source vertices of knn graph (size nnz)
   * @param cols destination vertices of knn graph (size nnz)
   * @param vals edge weights connecting vertices of knn graph (size nnz)
   * @param nnz size of rows/cols/vals
   * @param n number of samples in X
   * @param n_neighbors the number of neighbors to query for knn graph construction
   * @param n_components the number of components to project the X into
   * @param out output array for embedding (size n*n_comonents)
   */
void fit_embedding(const raft::handle_t &handle, int *rows, int *cols,
                   float *vals, int nnz, int n, int n_components, float *out,
                   unsigned long long seed) {
  raft::sparse::spectral::fit_embedding(handle, rows, cols, vals, nnz, n,
                                        n_components, out, seed);
}

void fit_embedding(const raft::handle_t &handle,
                   knn_indices_dense_t *knn_indices, float *knn_dists,
                   int n_components, float *out, uint64_t seed) {
  manifold_precomputed_knn_inputs_t<knn_indices_dense_t, float> inputs{
    knn_indices, knn_dists, X, nullptr, n, d, n_neighbors;
  };
  using value_t = float;
  using value_idx = int64_t;

  knn_graph<value_idx, value_t> knn_graph(inputs.n, k);

  knn_graph.knn_indices = knn_indices;
  knn_graph.knn_dists = knn_dists;

  kNNGraph::run<value_idx, value_t, manifold_precomputed_knn_inputs_t<knn_indices_dense_t, float>>(
    handle, inputs, inputs, knn_graph, k, params, d_alloc, stream);

  raft::sparse::spectral::fit_embedding(handle, rows, cols, vals, nnz, n,
                                        n_components, out, seed);
}
}  // namespace Spectral
}  // namespace ML
